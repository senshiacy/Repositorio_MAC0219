#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

//#define t 16
//#define b 16

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

#define BODY_TEMP 37
#define BODY_START_X 14
#define BODY_START_Y 30
#define BODY_END_X 34
#define BODY_END_Y 50

// Inicializador. Funciona.
void initialize(double *h, int n) {
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                h[i*n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else if (i >= BODY_START_X && i <= BODY_END_X && j >= BODY_START_Y && j <= BODY_END_Y) {
                h[i*n + j] = BODY_TEMP;
            } else {
                h[i*n + j] = 0.0;
            }
        }
    }

    return;
}

// Iteração de jacobi sequencial.
void jacobi_iteration(double *h, double *g, int n, int iter_limit) {
    for (int iter = 0; iter < iter_limit; iter++) {
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                if (!(i >= BODY_START_X && i <= BODY_END_X && j >= BODY_START_Y && j <= BODY_END_Y))
                g[i*n + j] = 0.25 * (h[(i-1)*n + j] + h[(i+1)*n + j] + h[i*n + j-1] + h[i*n + j+1]);
            }
        }
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                if (!(i >= BODY_START_X && i <= BODY_END_X && j >= BODY_START_Y && j <= BODY_END_Y))
                h[i*n + j] = g[i*n + j];
            }
        }
    }

    return;
}

// Kernel de jacobi
__global__ void jacobi_kernel(double *h, double *g, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (i < n-1 && j < n-1) {
        if (!(i >= BODY_START_X && i <= BODY_END_X && j >= BODY_START_Y && j <= BODY_END_Y))
        g[i*n + j] = 0.25 * (h[(i-1)*n +j] + h[(i+1)*n + j] + h[i*n + j-1] + h[i*n + j+1]);
    }

    return;
}

// Calcula tempo do sequencial
double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;

    return (end_sec - start_sec) / 1e9;
}

// Salva em arquivo o da CPU
void save_to_file_cpu(double *h, int n) {
    FILE *file = fopen("room-cpu.txt", "w");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(file, "%lf ", h[i*n + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);

    return;
}

// Salva em arquivo o da GPU
void save_to_file_gpu(double *h, int n) {
    FILE *file = fopen("room-gpu.txt", "w");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(file, "%lf ", h[i*n + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);

    return;
}

// Validador
void compareResults(double *cpu, double *gpu, int n) {
    double epsilon = 2*1e-2;
    for(int i = 0; i < n; i ++) {
        if (abs(cpu[i] - gpu[i]) > epsilon) {
            printf("Comparação de resultados falhou\n");
            exit(-1);
        }
    }
    printf("Comparação de resultados passou\n");
}


int main(int argc, char *argv[]) {
    if (argc < 5) {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações> <número de thread> <número de blocos>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int t = atoi(argv[3]);
    int b = atoi(argv[4]);
    // ^^ Comentar depois

    // Aloca memória necessária
    double *h_gpu; hipMalloc(&h_gpu, n*n * sizeof(double)); 
    double *g_gpu; hipMalloc(&g_gpu, n*n * sizeof(double));
    double *h_resposta = (double *)malloc(n*n * sizeof(double));
    double *h_cpu = (double *)malloc(n*n * sizeof(double));
    double *g_cpu = (double *)malloc(n*n * sizeof(double));

    if (h_resposta == NULL || h_gpu == NULL || g_gpu == NULL || h_cpu == NULL || g_cpu == NULL) {
        fprintf(stderr, "Erro ao alocar memória para h_resposta, h_gpu, g_gpu, h_cpu ou g_cpu\n");
        exit(EXIT_FAILURE);
    }

    // Inicializa o vetor
    struct timespec start_time, end_time;
    initialize(h_resposta, n);
    memcpy(h_cpu, h_resposta, (n*n * sizeof(double)));         // Copiando o vetor do h_resposta para cpu_h



    /************************************** CPU *********************************************/
        clock_gettime(CLOCK_MONOTONIC, &start_time);
        jacobi_iteration(h_cpu, g_cpu, n, iter_limit);
        clock_gettime(CLOCK_MONOTONIC, &end_time);

        double elapsed_time = calculate_elapsed_time(start_time, end_time);
        printf("Dados referentes à execução sequencial (CPU):\n");
        printf("Tempo de execução: %.9f ms\n", elapsed_time*1000);
    /****************************************************************************************/



    /************************************** GPU *********************************************/
        // Cria os eventos do CUDA para podermos ver o tempo de transferência
        hipEvent_t start, stop, copy_start, copy_stop;
        hipEventCreate(&start); hipEventCreate(&stop); hipEventCreate(&copy_start); hipEventCreate(&copy_stop);

        // Coloca em time_host_to_device o tempo em ms da cópia do host pra gpu
        hipEventRecord(copy_start);
        hipMemcpy(h_gpu, h_resposta, (n*n * sizeof(double)), hipMemcpyHostToDevice);
        hipMemcpy(g_gpu, h_resposta, (n*n * sizeof(double)), hipMemcpyHostToDevice);
        hipEventRecord(copy_stop);
        hipEventSynchronize(copy_stop);
        float time_host_to_device;
        hipEventElapsedTime(&time_host_to_device, copy_start, copy_stop);


        ///////////////// -----------------------

        // PARTE IMPORTANTE: OS CÁLCULOS!!
        // 1.1. Guarda o começo
        hipEventRecord(start);

        // 2. Faz as contas de fato
        dim3 threadsPerBlock(t,t);
        dim3 blocksPerGrid(b,b);
        for (int iter = 0; iter < iter_limit; iter++) {
            jacobi_kernel<<<blocksPerGrid, threadsPerBlock>>>(h_gpu, g_gpu, n);
            hipDeviceSynchronize();

            double *temp = h_gpu;
            h_gpu = g_gpu;
            g_gpu = temp;
        }

        // 1.2. Coloca em time_host_to_device o tempo de execução do programa
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float gpu_time;
        hipEventElapsedTime(&gpu_time, start, stop);

        ///////////////// -----------------------

        // Coloca em time_device_to_host o tempo em ms da cópia da gpu pra host
        hipEventRecord(copy_start);
        hipMemcpy(h_resposta, h_gpu, (n*n * sizeof(double)), hipMemcpyDeviceToHost);
        hipEventRecord(copy_stop);
        hipEventSynchronize(copy_stop);
        float time_device_to_host;
        hipEventElapsedTime(&time_device_to_host, copy_start, copy_stop);

        // Imprime os coisos
        printf("\nDados referentes à execução paralela (GPU):\n");
        printf("Tempo de GPU: %.9f ms\n", gpu_time);
        printf("Tempo de movimentação Host->Device: %.9f ms\n", time_host_to_device);
        printf("Tempo de movimentação Device->Host: %.9f ms\n\n", time_device_to_host);
    /****************************************************************************************/

    save_to_file_cpu(h_cpu, n);
    save_to_file_gpu(h_resposta, n);
    compareResults(h_cpu, h_resposta, n*n);

    free(h_resposta);
    free(h_cpu);
    free(g_cpu);
    hipFree(h_gpu);
    hipFree(g_gpu);

    return 0;
}